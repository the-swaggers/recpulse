#include "hip/hip_runtime.h"
#include "tensor.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>


void debug_log(const char* message) {
    FILE* log_file = fopen("/home/drexon/projects/recpulse/cuda_utils/tensor_debug.log", "a");
    if (log_file == NULL) {
        fprintf(stderr, "Could not open log file. Error message: %s\n", message);
        return;
    }
    fprintf(log_file, "%s\n", message);
    fflush(log_file);
    fclose(log_file);
}


__global__ void fill_kernel_scalar(float* vals, float value, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        vals[idx] = value;
    }
}

__global__ void fill_kernel_vals(float* vals, float* new_vals, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        vals[idx] = new_vals[idx];
    }
}


Tensor32* create_tensor(int* shape, int ndim) {
    Tensor32* tensor = (Tensor32*)malloc(sizeof(Tensor32));
    tensor->ndim = ndim;
    tensor->shape = (int*)malloc(ndim * sizeof(int));
    
    size_t size = 1;
    for (int i = 0; i < ndim; i++) {
        tensor->shape[i] = shape[i];
        size *= shape[i];
    }
    tensor->size = size;
    
    hipMalloc(&tensor->vals, size * sizeof(float));
    return tensor;
}

void free_tensor(Tensor32* tensor) {
    hipFree(tensor->vals);
    free(tensor->shape);
    free(tensor);
}

void fill_tensor_scalar(Tensor32* tensor, float value) {
    int block_size = 256;
    int num_blocks = (tensor->size + block_size - 1) / block_size;
    
    fill_kernel_scalar<<<num_blocks, block_size>>>(tensor->vals, value, tensor->size);
    hipDeviceSynchronize();
}

void fill_tensor_vals(Tensor32* tensor, float* new_vals) {
    int block_size = 256;
    int num_blocks = (tensor->size + block_size - 1) / block_size;
    
    fill_kernel_vals<<<num_blocks, block_size>>>(tensor->vals, new_vals, tensor->size);
    hipDeviceSynchronize();
}

void vals_from_tensor(Tensor32* tensor, float* array) {
    hipMemcpy(array, tensor->vals, tensor->size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize(); 
}

float element_from_tensor(Tensor32* tensor, int* idx) {
    int index = 0;
    int n = 1;
    for (int i = tensor->ndim - 1; i >= 0; i--) {
        index += n * idx[i];
        n *= tensor->shape[i];
    }
    float result;
    
    hipError_t err = hipMemcpy(&result, &(tensor->vals[index]), sizeof(float), hipMemcpyDeviceToHost);
    
    return result;
}

void free_array(float* arr) {
    free(arr);
}

//  Tensor32 cut_tensor(Tensor32* tensor, int* shape, int* slice) {
//      
//  }

